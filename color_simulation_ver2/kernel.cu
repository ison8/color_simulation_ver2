#define _CRT_SECURE_NO_WARNINGS

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>
#include <iomanip>
#include <vector>

#define D65_ROW 531		// D65の行数
#define D65_COL 2		// D65の列数
#define OBS_ROW 441		// 標準観測者の行数
#define OBS_COL 4		// 標準観測者の列数
#define XYZ_ROW 471		// xyzの行数
#define XYZ_COL 4		// xyzの列数
#define DATA_ROW 441	// 計算で使用するデータの行数 (390 - 830 nm)
#define DATA_MIN 390	// 使用する周波数の最小値
#define DATA_MAX 830	// 使用する周波数の最大値
#define PI 3.141592		// 円周率

#define BLOCKSIZE 441		// 1ブロック当たりのスレッド数
#define DATANUM 50			// 計算する数
#define CALCNUM 25000		// べき乗する数
#define SIMNUM 1000			// シミュレーションする回数
#define LOOPNUM 10			// SIMNUM回のシミュレーション繰り返す回数
#define GAUSS_CNT 10        // 足し合わせるガウシアンの数
#define GAUSS_PER 3         // ガウシアンのパラメータ数

#define MU_MIN  390         // μの最小値
#define MU_MAX  830         // μの最大値

using namespace std;

/* CUDAエラーチェック */
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

/* ファイルからデータを読み込む関数 
   d65, CIE LMS, CIE xyz を読み込む*/
int getFileData(double* d65, 
                double* obs_l, double* obs_m, double* obs_s,
                double* obs_x, double* obs_y, double* obs_z) {
    
    /* ファイルポインタ */
    FILE* fp_d65, * fp_obs, * fp_xyz;
    /* EOFを検出する変数 */
    int ret;
    /* カウンター */
    int count = 0;

    /*********************************************************************/
    /* D65の読み込み */
    /* ファイルオープン */
    fp_d65 = fopen("./d65.csv", "r");
    /* 正しく開けているかをチェック */
    if (fp_d65 == NULL) {
        cout << "File open error" << endl;
        return -1;
    }

    for (int i = 0; i < D65_ROW; i++) {
        /* 1時的に波長とデータを格納する変数 */
        double tmp_spt = 0, tmp_data = 0;
        /* 1行ずつ読み込む */
        ret = fscanf(fp_d65, "%lf, %lf", &tmp_spt, &tmp_data);
        /* 終了判定 */
        if (tmp_spt > DATA_MAX) {
            break;
        }
        /* カウンタ更新 */
        if (tmp_spt >= DATA_MIN) {
            d65[count] = tmp_data;
            count++;
        }
        /* エラーを検出した際の処理 */
        if (ret == EOF) {
            cout << "error" << endl;
            return -1;
        }
    }
    fclose(fp_d65);
    count = 0;
    /*********************************************************************/


    /*********************************************************************/
    /* 標準観測者(CIE LMS)の読み込み */
    /* ファイルオープン */
    fp_obs = fopen("./std_obs_10deg.csv", "r");
    /* 正しく開けているかをチェック */
    if (fp_obs == NULL) {
        cout << "File open error" << endl;
        return -1;
    }

    /* ファイル読み込み */
    for (int i = 0; i < OBS_ROW; i++) {
        /* 1時的に波長とデータを格納する変数 */
        double tmp_spt = 0, tmp_l = 0, tmp_m = 0, tmp_s = 0;
        /* 1行ずつ読み込む */
        ret = fscanf(fp_obs, "%lf, %lf, %lf, %lf", &tmp_spt, &tmp_l, &tmp_m, &tmp_s);
        /* 終了判定 */
        if (tmp_spt > DATA_MAX) {
            break;
        }
        /* カウンタの更新 */
        if (tmp_spt >= DATA_MIN) {
            obs_l[count] = tmp_l;
            obs_m[count] = tmp_m;
            obs_s[count] = tmp_s;
            count++;
        }
        /* エラーを検出した際の処理 */
        if (ret == EOF) {
            cout << "error" << endl;
            return -1;
        }
    }
    fclose(fp_obs);
    count = 0;
    /*********************************************************************/
    

    /*********************************************************************/
    /* xyzの読み込み */
    /* ファイルオープン */
    fp_xyz = fopen("./ciexyz31.csv", "r");
    /* 正しく開けているかをチェック */
    if (fp_xyz == NULL) {
        cout << "File open error" << endl;
        return -1;
    }
    /* ファイル読み込み */
    for (int i = 0; i < XYZ_ROW; i++) {
        /* 1時的に波長とデータを格納する変数 */
        double tmp_spt = 0, tmp_x = 0, tmp_y = 0, tmp_z = 0;
        /* 1行ずつ読み込む */
        ret = fscanf(fp_xyz, "%lf, %lf, %lf, %lf", &tmp_spt, &tmp_x, &tmp_y, &tmp_z);
        /* 終了判定 */
        if (tmp_spt > DATA_MAX) {
            break;
        }
        /* カウンタの更新 */
        if (tmp_spt >= DATA_MIN) {
            obs_x[count] = tmp_x;
            obs_y[count] = tmp_y;
            obs_z[count] = tmp_z;
            count++;
        }
        /* エラーを検出した際の処理 */
        if (ret == EOF) {
            cout << "error" << endl;
            return -1;
        }
    }
    fclose(fp_xyz);

    return 0;
    /*********************************************************************/
}

/* ガウシアン生成 */
void calcGauss(double* gauss_data) {
    for (int i = 0; i < (SIMNUM * LOOPNUM * GAUSS_CNT * GAUSS_PER); i += 3) {
        /* μ */
        double mu = MU_MIN + ((double)rand() / RAND_MAX * (double)(MU_MAX - MU_MIN));
        /* σ */
        double sigma = 5 + (95 * (double)rand() / RAND_MAX);
        /* 振幅の最大値 */
        double g_amp = (double)rand() / RAND_MAX;
        /* gauss_dataに格納 */
        gauss_data[i] = mu;
        gauss_data[i + 1] = sigma;
        gauss_data[i + 2] = g_amp;
    }
}

int main(void) {
    /* データを入れる１次元配列 */
    double* d65, * obs_x, * obs_y, * obs_z, * obs_l, * obs_m, * obs_s, * gauss_data, * result, * fin_result, * lms_result, * lms_fin;
    int* mesh_result, * mesh_f_result;
    /* 配列のメモリ確保 */
    d65 = new double[DATA_ROW];
    obs_l = new double[DATA_ROW];
    obs_m = new double[DATA_ROW];
    obs_s = new double[DATA_ROW];
    obs_x = new double[DATA_ROW];
    obs_y = new double[DATA_ROW];
    obs_z = new double[DATA_ROW];
    gauss_data = new double[SIMNUM * LOOPNUM * GAUSS_CNT * GAUSS_PER];

    /* ファイル読み込み関数実行 */
    int f_result = getFileData(d65, obs_l, obs_m, obs_s, obs_x, obs_y, obs_z);

    /* ガウシアンの要素を生成 */
    calcGauss(gauss_data);

    /* 出力ディレクトリ */
    string directory = "C:/Users/KoidaLab-WorkStation/Desktop/isomura_ws/color_simulation_result/sim_1000_10000_10_v1/";
    string fname = "test.csv";
    string o_fname = directory + fname;

    /* ファイル出力ストリーム */
    ofstream o_file(o_fname);

    /* ファイル出力 */
    for (int i = 0; i < (SIMNUM * LOOPNUM * GAUSS_CNT * GAUSS_PER); i += 3) {
        o_file << gauss_data[i] << "," << gauss_data[i+1] << "," << gauss_data[i+2] << endl;
    }
    return 0;
}